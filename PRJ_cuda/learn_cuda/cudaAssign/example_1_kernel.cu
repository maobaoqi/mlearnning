#include "hip/hip_runtime.h"
﻿#ifndef EXAMPLE_1_KERNEL_H_
#define EXAMPLE_1_KERNEL_H_

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <cuda_occupancy.h>
#include <device_types.h>
#include <hip/device_functions.h>
#include <device_atomic_functions.hpp>
#include <stdlib.h>
#include <stdio.h>
#include <curt>

__global__ void testKernel(float* g_data_in, float* g_data_out)
{
	extern __shared__ float sdata[];

	const unsigned int blockID = blockIdx.x;
	const unsigned int threadID_in_block = threadIdx.x;
	const unsigned int threadID_in_grid = blockDim.x * blockIdx.x + threadIdx.x;

	sdata[threadID_in_block] = g_data_in[threadID_in_grid];
	__syncthreads();
}


// 简单的规约求和函数
__global__ static void reduceKernel(float* d_result, float* d_input, int N)
{
	const int threadID = blockDim.x * blockIdx.x + threadIdx.x;
	const int threadN = gridDim.x * blockDim.x;

	float sum = 0;
	for (int pos = threadID; pos < N; pos += threadN)
	{
		sum += d_input[pos];
	}

	d_result[threadID] = sum;
}

// 将内核函数封装成C函数
extern "C"
void lanunch_reduceKernel(float* d_result, float* d_input, int N, int BLOCK_N, int THREAD_N)
{
	reduceKernel <<<BLOCK_N, THREAD_N >>> (d_result, d_input, N);
}

#endif