#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <IntShCut.h>

void runTest(int argc, char** agrv);

int main(int argc, char** argv)
{
	runTest(argc, argv);
	
	return 0;
}

void runTest(int argc, char** argv)
{
	// start
	hipSetDevice(0);

	unsigned int num_blocks = 4;
	unsigned int num_threads = 4;

	unsigned int mem_size = sizeof(float)*num_threads*num_blocks;
	float* h_data_in = (float*)malloc(mem_size);
	float* h_data_out = (float*)malloc(mem_size);

	float* d_data_in;
	hipMalloc((void**)&d_data_in, mem_size);
	float* d_data_out;
	hipMalloc((void**)&d_data_out, mem_size);

	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);

	// run kernel

	// call back
	hipMemcpy(h_data_out, d_data_out, mem_size, hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < num_blocks; i++)
	{
		for (unsigned int j = 0; j < num_threads; j++)
		{
			printf("%5.0f", h_data_out[i*num_threads + j]);
		}
		printf("\n");
	}

	free(h_data_in);
	free(h_data_out);
	hipFree(d_data_in);
	hipFree(d_data_out);
}