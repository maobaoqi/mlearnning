// kernel definition
#include "hip/hip_runtime.h"


__global__ void vecAdd(float* A, float* B, float* C)
{
	int idx = threadIdx.x;
	C[idx] = A[idx] + B[idx];
}

//int main(void)
//{
//
//	//vecAdd << <1, N >> > (A, B, C);
//}